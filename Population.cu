#include <iostream>
#include <iomanip>
#include <stdlib.h>
#include <time.h>
#include <fstream>
#include "Fitness.h"
#include "CudaEval.h"

#define POPULATION_SIZE 150
#define POPULATION_PADDING 1000
#define CHROMOSOME_LENGTH 903
#define CROSSES 100000
#define CROSSOVER_FUNCTIONS 3
#define START_CLIMBING 5000

typedef struct member_struct {
    char* chromosome;
    int num_cliques;
} MEMBER;

void PrintPopulation(MEMBER[]);
void SortInitialPopulation(MEMBER[], int, int);
void Cross(MEMBER*, MEMBER*, MEMBER*);
void InitializeRandomMember(MEMBER*);
void PrintMatrix(int[N][N]);
void InsertMember(MEMBER[], MEMBER);
void BiasedCross(MEMBER[2], MEMBER*);
void BiasederCross(MEMBER[2], MEMBER*);
void RandomSinglePointCross(MEMBER[2], MEMBER*);
void Mutate(MEMBER*);
void Climb(MEMBER*);
int EvalAdj(char[N][N]);

int main(int argc, const char* argv[])
{
	std::ofstream file;
	file.open("ramsey.txt", std::ios::app);
	
	/* init cross pointers */
	void (*Cross[CROSSOVER_FUNCTIONS])(MEMBER[2], MEMBER*) = {NULL};
	Cross[0] = &BiasedCross;
	Cross[1] = &BiasederCross;
	Cross[2] = &RandomSinglePointCross;
	
	unsigned int seed = time(NULL);
    //unsigned int seed = 1367392616;
	srand(seed); //init random seed
	CudaInit();

	std::cout << "INITIALIZING POPULATION" << std::endl;

    MEMBER population[POPULATION_SIZE];

    for (int i = 0; i < POPULATION_SIZE; i++) {
        InitializeRandomMember(&population[i]);
    }

	SortInitialPopulation(population, 0, POPULATION_SIZE - 1);
	PrintPopulation(population);

	std::cout << "PADDING POPULATION" << std::endl << std::endl;

	for (int i = 0; i < POPULATION_PADDING; i++) {
		MEMBER member;
		InitializeRandomMember(&member);
		InsertMember(population, member);
	}
	PrintPopulation(population);

	std::cout << "BREEDING" << std::endl << std::endl;

	/* breed children */
	int best = 999999;
	for (int i = 0; i < CROSSES; i++) {
		
		if (i > START_CLIMBING && i % 500 == 0) {
			std::cout << "CLIMBING" << std::endl;
			for (int j = 0; j < POPULATION_SIZE; j++) {
				Climb(&population[j]);
			}
			SortInitialPopulation(population, 0, POPULATION_SIZE - 1);
			if (population[0].num_cliques < best) {
				best = population[0].num_cliques;
				std::cout << "Current best (H): " << best << std::endl;
			}
		}

		MEMBER parents[2];
		MEMBER child;

		parents[0] = population[rand() % POPULATION_SIZE];
		parents[1] = population[rand() % POPULATION_SIZE];
		(*Cross[0])(parents, &child);

		/*parents[0] = population[rand() % (POPULATION_SIZE / 2)];
		//parents[1] = population[(rand() % (POPULATION_SIZE / 2) + POPULATION_SIZE / 2)];
		//(*Cross[1])(parents, &child);
		//(*Cross[0])(parents, &child);

		/* NOT GOING TO HAPPEN */
		if (child.num_cliques < 10) {
			std::cout << child.num_cliques << ":" << std::endl;

			for (int j = 0; j < CHROMOSOME_LENGTH; j++) {
				std::cout << (char) (child.chromosome[j] + 0x30);
			}

			std::cout << std::endl;
		}

		InsertMember(population, child);

		if (population[0].num_cliques < best) {
			best = population[0].num_cliques;
			std::cout << "Current best (X): " << best << std::endl;
		}

		if (population[POPULATION_SIZE - 1].num_cliques == population[0].num_cliques) {
			std::cout << "MIGRATING" << std::endl;
			for (int j = 5; j < POPULATION_SIZE; j++) {
				free(population[i].chromosome);
				InitializeRandomMember(&population[i]);
			}
		}
	}

	std::cout << std::endl;
	PrintPopulation(population);
	std::cout << "Best member: " << population[0].num_cliques << std::endl;
	for (int j = 0; j < CHROMOSOME_LENGTH; j++) {
		std::cout << (char) (population[0].chromosome[j] + 0x30);
	}

	std::cout << std::endl;

	/* echo seed for posterity */
	std::cout << "SEED: " << seed << std::endl;

	file << "BEST: " << best << std::endl;
	file << "ENCODING: " << std::endl;
	for (int j = 0; j < CHROMOSOME_LENGTH; j++) {
		file << (char) (population[0].chromosome[j] + 0x30);
	}
	file << std::endl;
	file << "SEED: " << seed << std::endl << std::endl;
	file.close();

    /* leave console up until keypress */
	std::cout << "FINISHED AND WAITING FOR RETURN KEY" << std::endl;
    std::getchar();
}

/*
 * Prints an array of MEMBER.
 */
void PrintPopulation(MEMBER population[])
{
    for (int i = 0; i < POPULATION_SIZE; i++) {
        std::cout << "Member " << std::setw(3) << i + 1 << ": " ;
        std::cout << std::setw(4) << population[i].num_cliques << std::endl;
    }
    std::cout << std::endl;
}

/*
 * Performs a quicksort on an array of MEMBER.
 */
void SortInitialPopulation(MEMBER population[], int left, int right)
{
    int i = left;
    int j = right;
    int pivot = population[(left + right) / 2].num_cliques;
    MEMBER temp;

    /* partition */
    while (i <= j) {
        while (population[i].num_cliques < pivot) {
            i++;
        }
        while (population[j].num_cliques > pivot) {
            j--;
        }
        if (i <= j) {
            temp = population[i];
            population[i] = population[j];
            population[j] = temp;
            i++;
            j--;
        }
    };

    /* recursively sort either side of pivot */
    if (left < j) {
        SortInitialPopulation(population, left, j);
    }
    if (i < right) {
        SortInitialPopulation(population, i, right);
    }

    return;
}

void InsertMember(MEMBER population[], MEMBER member)
{
	if (member.num_cliques < population[POPULATION_SIZE - 1].num_cliques) {
		free(population[POPULATION_SIZE - 1].chromosome);
		population[POPULATION_SIZE - 1] = member;
		for (int i = POPULATION_SIZE - 1; i > 0; i--) {
			if (population[i].num_cliques < population[i - 1].num_cliques) {
				population[i] = population[i - 1];
				population[i - 1] = member;
			} else {
				break;
			}
		}
	} else {
		free(member.chromosome);
	}
}

void InitializeRandomMember(MEMBER *member)
{
    char *child_chromosome = (char*)(malloc(sizeof(char) * CHROMOSOME_LENGTH));
    for (int i = 0; i < CHROMOSOME_LENGTH; i++) {
        child_chromosome[i] = rand() % 2;
    }

	char adjacency_matrix[N][N];
    GetAdjacencyMatrixFromCharArray(child_chromosome, adjacency_matrix);
    int num_cliques = EvalAdj(adjacency_matrix);
    member->chromosome = child_chromosome;
    member->num_cliques = num_cliques;
}

void PrintMatrix(int arr[N][N]) {
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			std::cout << arr[i][j] << " ";
		}
		std::cout << std::endl;
	}
}

void Mutate(MEMBER *member) {
	int bit = rand() % CHROMOSOME_LENGTH;
	member->chromosome[bit] ^= 1;
}

void Climb(MEMBER *member) {
	char *original_chromosome = member->chromosome;
	char *new_chromosome = (char*)(malloc(sizeof(char) * CHROMOSOME_LENGTH));

	for (int i = 0; i < CHROMOSOME_LENGTH; i++) {
		new_chromosome[i] = original_chromosome[i];
	}

	int bit = rand() % CHROMOSOME_LENGTH;
	new_chromosome[bit] ^= 1;

	char adjacency_matrix[N][N];
    GetAdjacencyMatrixFromCharArray(new_chromosome, adjacency_matrix);
    int num_cliques = EvalAdj(adjacency_matrix);

	if (num_cliques < member->num_cliques) {
		member->num_cliques = num_cliques;
		free(member->chromosome);
		member->chromosome = new_chromosome;
	} else {
		free(new_chromosome);
	}
}

void BiasedCross(MEMBER parents[2], MEMBER *child)
{
    char *child_chromosome = (char*)(malloc(sizeof(char) * CHROMOSOME_LENGTH));

	float bias;
	float parent_cliques[2];
	parent_cliques[0] = (float)parents[0].num_cliques;
	parent_cliques[1] = (float)parents[1].num_cliques;

	bias = parent_cliques[parent_cliques[0] < parent_cliques[1]] / (parent_cliques[0] + parent_cliques[1]);
	MEMBER bad;
	MEMBER good;
	if (parent_cliques[0] < parent_cliques[1]) {
		bad = parents[1];
		good = parents[0];
	} else {
		bad = parents[0];
		good = parents[1];
	}

	for (int i = 0; i < CHROMOSOME_LENGTH; i++) {
		if (((float)rand() / (float)RAND_MAX) > bias) {
			child_chromosome[i] = bad.chromosome[i];
		} else {
			child_chromosome[i] = good.chromosome[i];
		}
	}

	char adjacency_matrix[N][N];
    GetAdjacencyMatrixFromCharArray(child_chromosome, adjacency_matrix);
    int num_cliques = EvalAdj(adjacency_matrix);

    child->chromosome = child_chromosome;
    child->num_cliques = num_cliques;
}

void BiasederCross(MEMBER parents[2], MEMBER *child)
{
	char *chromosome[2];
    chromosome[0] = parents[0].chromosome;
    chromosome[1] = parents[1].chromosome;

    char *child_chromosome = (char*)(malloc(sizeof(char) * CHROMOSOME_LENGTH));

	float bias;
	float parent_cliques[2];
	parent_cliques[0] = (float)parents[0].num_cliques;
	parent_cliques[1] = (float)parents[1].num_cliques;

	bias = parent_cliques[1] / (parent_cliques[0] + parent_cliques[1]);

	for (int i = 0; i < CHROMOSOME_LENGTH; i++) {
		if (((float)rand() / (float)RAND_MAX) > bias) {
			child_chromosome[i] = chromosome[1][i];
		} else {
			child_chromosome[i] = chromosome[0][i];
		}
	}

	char adjacency_matrix[N][N];
    GetAdjacencyMatrixFromCharArray(child_chromosome, adjacency_matrix);
    int num_cliques = EvalAdj(adjacency_matrix);

    child->chromosome = child_chromosome;
    child->num_cliques = num_cliques;
}

void RandomSinglePointCross(MEMBER parents[2], MEMBER *child)
{
	char *chromosome[2];
    chromosome[0] = parents[0].chromosome;
    chromosome[1] = parents[1].chromosome;

    char *child_chromosome = (char*)(malloc(sizeof(char) * CHROMOSOME_LENGTH));

    int crossover = rand() % CHROMOSOME_LENGTH;
    
	for (int i = 0; i < crossover; i++) {
        child_chromosome[i] = chromosome[0][i];
    }

    for (int i = crossover; i < CHROMOSOME_LENGTH; i++) {
        child_chromosome[i] = chromosome[1][i];
    }

	char adjacency_matrix[N][N];
    GetAdjacencyMatrixFromCharArray(child_chromosome, adjacency_matrix);
    int num_cliques = EvalAdj(adjacency_matrix);

    child->chromosome = child_chromosome;
    child->num_cliques = num_cliques;
}

int EvalAdj(char adj[N][N]) {
	return CudaEval((char *) adj);
}

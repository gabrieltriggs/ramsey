#include <iostream>
#include <iomanip>
#include <stdlib.h>
#include <time.h>
#include <fstream>
#include "Fitness.h"
#include "CudaEval.h"

#define POPULATION_SIZE 200 // number of members in a population
#define POPULATION_PADDING 1500 // number of extra members to make for padding
#define CHROMOSOME_LENGTH 903 // length of bitstring representation of graph
#define CROSSES 200000 // number of crosses to complete before starting over
#define START_CLIMBING 5000 // number of crosses to complete before climbing
#define START_MUTATION 50000 // number of crosses to complete before mutating
#define MUTATION_PERCENTAGE .1 // percent of bits to flip during mutation
#define CROSSOVER_FUNCTIONS 2 // number of crossovers
#define CROSSOVER_RANDOMIZATION_POINT 300 // score at which to randomize crossover

/* Single member of the population. Contains bitstring and score */
typedef struct member_struct {
    char* chromosome;
    int num_cliques;
} MEMBER;

void InitializePopulation(MEMBER[]);
void PadPopulation(MEMBER[]);
void PrintPopulation(MEMBER[]);
void QuicksortPopulation(MEMBER[], int, int);
void Cross(MEMBER*, MEMBER*, MEMBER*);
void InitializeRandomMember(MEMBER*);
void InsertMemberIntoPopulation(MEMBER, MEMBER[]);
void CrossWithBias(MEMBER[2], MEMBER*);
void CrossAtRandomSinglePoint(MEMBER[2], MEMBER*);
void Breed(MEMBER[], void (*Cross[CROSSOVER_FUNCTIONS])(MEMBER[2], MEMBER*));
void Mutate(MEMBER*);
void Climb(MEMBER*);
int EvaluateAdjacencyMatrix(char[N][N]);


int main(int argc, const char* argv[])
{	
	/* init crossover pointers */
	void (*Cross[CROSSOVER_FUNCTIONS])(MEMBER[2], MEMBER*) = {NULL};
	Cross[0] = &CrossWithBias;
	Cross[1] = &CrossAtRandomSinglePoint;
	
	/* init random seed */
	unsigned int seed = time(NULL);
	srand(seed);

	/* init cache of subsets on device */
	CudaInit();

	/* loops entire algorithm until halted */
	while (1) {

		MEMBER population[POPULATION_SIZE];

		/* initialize, pad, and print population */
		InitializePopulation(population);
		PadPopulation(population);
		PrintPopulation(population);

		/* do the work of actually breeding population */
		Breed(population, Cross);

		std::ofstream file;
		file.open("ramsey.txt", std::ios::app);
		file << "BEST: " << population[0].num_cliques << std::endl;
		file << "ENCODING: " << std::endl;
		for (int j = 0; j < CHROMOSOME_LENGTH; j++) {
			file << (char) (population[0].chromosome[j] + 0x30);
		}
		file << std::endl;
		file << "SEED: " << seed << std::endl << std::endl;
		file.close();

		for (int z = 0; z < POPULATION_SIZE; z++) {
			free(population[z].chromosome);
		}
	}
}

/*
 * Sets each space in a population array to a new random member
 */
void InitializePopulation(MEMBER population[])
{
	std::cout << "INITIALIZING POPULATION" << std::endl;
	for (int i = 0; i < POPULATION_SIZE; i++) {
		InitializeRandomMember(&population[i]);
	}
	QuicksortPopulation(population, 0, POPULATION_SIZE - 1);
}

/*
 * Pads population with ramdom members in hopes of getting
 * a more fit pool with which to begin.
 */
void PadPopulation(MEMBER population[])
{
	std::cout << "PADDING POPULATION" << std::endl << std::endl;

	for (int i = 0; i < POPULATION_PADDING; i++) {
		MEMBER member;
		InitializeRandomMember(&member);
		InsertMemberIntoPopulation(member, population);
	}
}

/*
 * Prints scores for an array of MEMBER.
 */
void PrintPopulation(MEMBER population[])
{
    for (int i = 0; i < POPULATION_SIZE; i++) {
        std::cout << "Member " << std::setw(3) << i + 1 << ": " ;
        std::cout << std::setw(4) << population[i].num_cliques << std::endl;
    }
    std::cout << std::endl;
}

/*
 * Performs a quicksort on an array of MEMBER.
 */
void QuicksortPopulation(MEMBER population[], int left, int right)
{
    int i = left;
    int j = right;
    int pivot = population[(left + right) / 2].num_cliques;
    MEMBER temp;

    /* partition */
    while (i <= j) {
        while (population[i].num_cliques < pivot) {
            i++;
        }
        while (population[j].num_cliques > pivot) {
            j--;
        }
        if (i <= j) {
            temp = population[i];
            population[i] = population[j];
            population[j] = temp;
            i++;
            j--;
        }
    };

    /* recursively sort either side of pivot */
    if (left < j) {
        QuicksortPopulation(population, left, j);
    }
    if (i < right) {
        QuicksortPopulation(population, i, right);
    }

    return;
}

void InsertMemberIntoPopulation(MEMBER member, MEMBER population[])
{
	if (member.num_cliques < population[POPULATION_SIZE - 1].num_cliques) {
		free(population[POPULATION_SIZE - 1].chromosome);
		population[POPULATION_SIZE - 1] = member;
		for (int i = POPULATION_SIZE - 1; i > 0; i--) {
			if (population[i].num_cliques < population[i - 1].num_cliques) {
				population[i] = population[i - 1];
				population[i - 1] = member;
			} else {
				break;
			}
		}
	} else {
		free(member.chromosome);
	}
}

void InitializeRandomMember(MEMBER *member)
{
    char *child_chromosome = (char*)(malloc(sizeof(char) * CHROMOSOME_LENGTH));
    for (int i = 0; i < CHROMOSOME_LENGTH; i++) {
        child_chromosome[i] = rand() % 2;
    }

	char adjacency_matrix[N][N];
    GetAdjacencyMatrixFromCharArray(child_chromosome, adjacency_matrix);
    int num_cliques = EvaluateAdjacencyMatrix(adjacency_matrix);
    member->chromosome = child_chromosome;
    member->num_cliques = num_cliques;
}

void Breed(MEMBER population[], void (*Cross[CROSSOVER_FUNCTIONS])(MEMBER[2], MEMBER*))
{
	std::cout << "BREEDING" << std::endl << std::endl;

	/* breed children */
	int best = 999999;
	for (int i = 0; i < CROSSES; i++) {
		if (i > START_CLIMBING && i % 500 == 0) {
			std::cout << "CLIMBING" << std::endl;
			for (int j = 0; j < POPULATION_SIZE; j++) {
				Climb(&population[j]);
			}
			QuicksortPopulation(population, 0, POPULATION_SIZE - 1);
			if (population[0].num_cliques < best) {
				best = population[0].num_cliques;
				std::cout << "Current best (H): " << best << std::endl;
			}
		}
		if (i > START_MUTATION && i % 2000 == 0) {
			std::cout << "MUTATING" << std::endl;
			int x;
			for (int j = 0; j < (int) ((float) POPULATION_SIZE * 0.25); j++) {
				x = rand() % (POPULATION_SIZE - 5) + 5;
				Mutate(&population[x]);
			}
		}
		
		MEMBER parents[2];
		MEMBER child;
		
		parents[0] = population[rand() % ((int) ((float) POPULATION_SIZE * 0.3))];
		parents[1] = population[rand() % (((int) ((float) POPULATION_SIZE * 0.7)) + ((int) ((float) POPULATION_SIZE * 0.3)))];
		
		int cross = population[0].num_cliques < CROSSOVER_RANDOMIZATION_POINT? rand() % 2 : 0;
		(*Cross[cross])(parents, &child);
		InsertMemberIntoPopulation(child, population);
		
		if (population[0].num_cliques < best) {
			best = population[0].num_cliques;
			std::cout << "Current best (X): " << best << std::endl;
		}

		/* NOT GOING TO HAPPEN */
		if (best < 10) {
			std::cout << population[0].num_cliques << ":" << std::endl;

			for (int j = 0; j < CHROMOSOME_LENGTH; j++) {
				std::cout << (char) (population[0].chromosome[j] + 0x30);
			}
			
			std::cout << std::endl;
		}

		if (population[POPULATION_SIZE - 1].num_cliques == population[0].num_cliques) {
			std::cout << "MIGRATING" << std::endl;
			for (int j = (int) ((float) POPULATION_SIZE * 0.05); j < POPULATION_SIZE; j++) {
				free(population[j].chromosome);
				InitializeRandomMember(&population[j]);
			}
			QuicksortPopulation(population, 0, POPULATION_SIZE - 1);
		}
	}

	std::cout << std::endl;
	PrintPopulation(population);
	std::cout << "Best member: " << population[0].num_cliques << std::endl;
	
	for (int j = 0; j < CHROMOSOME_LENGTH; j++) {
		std::cout << (char) (population[0].chromosome[j] + 0x30);
	}
}

void Mutate(MEMBER *member) {
	int bit;
	for (int i = 0; i < (int) CHROMOSOME_LENGTH * MUTATION_PERCENTAGE; i++) {
		bit = rand() % CHROMOSOME_LENGTH;
		member->chromosome[bit] ^= 1;
	}
}

void Climb(MEMBER *member) {
	
	int bit = rand() % CHROMOSOME_LENGTH;
	member->chromosome[bit] ^= 1;

	char adjacency_matrix[N][N];
    GetAdjacencyMatrixFromCharArray(member->chromosome, adjacency_matrix);
    int num_cliques = EvaluateAdjacencyMatrix(adjacency_matrix);

	if (num_cliques < member->num_cliques) {
		member->num_cliques = num_cliques;
	} else {
		member->chromosome[bit] ^= 1;
	}
}

void CrossWithBias(MEMBER parents[2], MEMBER *child)
{
    char *child_chromosome = (char*)(malloc(sizeof(char) * CHROMOSOME_LENGTH));

	float bias;
	float parent_cliques[2];
	parent_cliques[0] = (float)parents[0].num_cliques;
	parent_cliques[1] = (float)parents[1].num_cliques;

	bias = parent_cliques[parent_cliques[0] < parent_cliques[1]] / (parent_cliques[0] + parent_cliques[1]);
	MEMBER bad;
	MEMBER good;
	if (parent_cliques[0] < parent_cliques[1]) {
		bad = parents[1];
		good = parents[0];
	} else {
		bad = parents[0];
		good = parents[1];
	}

	for (int i = 0; i < CHROMOSOME_LENGTH; i++) {
		if (((float)rand() / (float)RAND_MAX) > bias) {
			child_chromosome[i] = bad.chromosome[i];
		} else {
			child_chromosome[i] = good.chromosome[i];
		}
	}

	char adjacency_matrix[N][N];
    GetAdjacencyMatrixFromCharArray(child_chromosome, adjacency_matrix);
    int num_cliques = EvaluateAdjacencyMatrix(adjacency_matrix);

    child->chromosome = child_chromosome;
    child->num_cliques = num_cliques;
}

void CrossAtRandomSinglePoint(MEMBER parents[2], MEMBER *child)
{
	/*char *chromosome[2];
    chromosome[0] = parents[0].chromosome;
    chromosome[1] = parents[1].chromosome;*/

    char *child_chromosome = (char*)(malloc(sizeof(char) * CHROMOSOME_LENGTH));
	char *child_chromosome2 = (char*)(malloc(sizeof(char) * CHROMOSOME_LENGTH));

    int crossover = rand() % CHROMOSOME_LENGTH;
    
	for (int i = 0; i < crossover; i++) {
        //child_chromosome[i] = chromosome[0][i];
		child_chromosome[i] = parents[0].chromosome[i];
		child_chromosome2[i] = parents[1].chromosome[i];
	}

    for (int i = crossover; i < CHROMOSOME_LENGTH; i++) {
        //child_chromosome[i] = chromosome[1][i];
		child_chromosome[i] = parents[1].chromosome[i];
		child_chromosome2[i] = parents[0].chromosome[i];
    }

	char adjacency_matrix[N][N];
    GetAdjacencyMatrixFromCharArray(child_chromosome, adjacency_matrix);
    int num_cliques = EvaluateAdjacencyMatrix(adjacency_matrix);

	char adjacency_matrix2[N][N];
	GetAdjacencyMatrixFromCharArray(child_chromosome2, adjacency_matrix2);
	int num_cliques2 = EvaluateAdjacencyMatrix(adjacency_matrix);

	if (num_cliques < num_cliques2) {
		child->chromosome = child_chromosome;
		child->num_cliques = num_cliques;
	} else {
		child->chromosome = child_chromosome2;
		child->num_cliques = num_cliques2;
	}
}

int EvaluateAdjacencyMatrix(char adj[N][N]) {
	return CudaEval((char *) adj);
}

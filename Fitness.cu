#include "hip/hip_runtime.h"
/*Fitness function to evaluate the number of monochromatic 
 * cliques present in a given graph.
 *
 * Jon Johnson, Gabriel Triggs
 */
#include <iostream>
#include <stdlib.h>
#include <string>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#define N 43
#define K 5
#define KC2 10

//fuction declarations
int Choose(int, int);
int EvaluateEdges(int[]);
void GetAdjacencyMatrixFromCharArray(char[], int[N][N]);
void GetEdgePossibilities(int[KC2][2]);
void GetElement(int, int[]);
void GetElement(int, int, int, int[]);
int GetLargestV(int, int, int);

//global variables
int choose_cache[N][K];
int edges[KC2][2];
std::string bit_string = "011001110000011100100001110110000000010100110010010111110111010100111111110100011001000010011001111100101011111000001010011101101110100101011001100110001101101000000011000010110100011111001110100010101011001110010001110000111101000101010111100100000101110111101101011010000011000110001101001110110110111001011001101011110100011111011010001100010100010100101011110101100010001100101111011011000010110000101001001010101000101110110111110011101100001000100011111011101001010101111010101110011010001111000110111010011011001001011001100000000100110010111111100010010001011010110011010101001110010100111001001011100100100100100010110011110000100101010111110101101000001111001111101100011111001010101000010011001110100100011100101011000011100010101110011101111000101000110001010100100111100101111011010100001100010010101011000100010101110101010111101001110000110110101001000010011110111001100101111011100001010";
//const char *char_bits = bit_string.c_str();
char *char_bits = new char[bit_string.size()];
int adjacency_matrix[N][N];

void print()
{
    std::cout << "test" << std::endl;
    std::cin >> "%d";
}

int main(int argc, char *argv[])
{
    /* init char_bits */
    for (int i = 0; i < bit_string.size(); i++) {
        char_bits[i] = (bit_string[i] == '0') ? 0 : 1;
    }

    //print();

    GetEdgePossibilities(edges);
    GetAdjacencyMatrixFromCharArray(char_bits, adjacency_matrix);

    //print();

    srand (time(NULL)); // init random seed
    int upper_bound = Choose(N, K);
    std::cout << upper_bound << std::endl;
    int arr[5] = { 0, 0, 0, 0, 0 };
    //print();

    // need to figure out time stuff
    //long total_time = 0;
    //long first_time = time(NULL); // get current time
    //long time_limit = 5;
    //print();

    do {
        int num_cliques = 0;

        /* evaluate every possible clique */
        for (int i = 0; i < upper_bound; i++) {
            GetElement(i, arr);
            
            int result = EvaluateEdges(arr);
            
            if (result == 0 || result == KC2) {
                num_cliques++;
                //print();
            }
        }

        //long current_time = time(NULL);
        //total_time = current_time - first_time;
        
        std::cout << "Cliques: " << num_cliques << std::endl;

        if (num_cliques < 10) {
            /* not going to happen */
            for (int j = 0; j < bit_string.size(); j++) {
                std::cout << char_bits[j];
            }
            std::cout << std::endl;
        }

        /* flip a random bit */
        int x = rand() % bit_string.size(); // random number over [0, char_bits.length)
        char_bits[x] = !char_bits[x];
        GetAdjacencyMatrixFromCharArray(char_bits, adjacency_matrix);
    } while (1); //FIX THIS FIX IT FIX ITASDFGFJASDFIFAJSDIFJASWDIFJASDIFJASCIFAJSIFAJSDKIFAHJSDIFNHJASDIFNHASDIFNHASDIFHASDIHFASKIDNHFASDKINHF
}

/*
 * Returns nCk.
 */
int Choose(int n, int k)
{
    if (n < k) {
        return 0;
    } else if (n == k) {
        return 1;
    }

    /* check to see whether it's already cached */
    if (choose_cache[n-1][k-1] != 0) {
        return choose_cache[n-1][k-1];
    }

    /* take advantage of the fact that nCk == nC(n-k) to do faster computation */
    int diff;
    int max;

    if (k < (n - k)) {
        diff = n - k;
        max = k;
    } else {
        diff = k;
        max = n - k;
    }

    int ans = diff + 1;

    for (int i = 2; i <= max; i++) {
        ans = (ans * (diff + i)) / i;
    }

    /* cache answer before returning */
    choose_cache[n-1][k-1] = ans;

    return ans;
}

/*
 * Returns the number of edges in arr that are "red" (1).
 *
 * If this return KC2, it is a red clique.
 * If this returns 0, it is a blue clique.
 */
int EvaluateEdges(int arr[])
{
    int result = 0;

    /*for (int i = 0; i < KC2; i++) {
        result += adjacency_matrix[arr[edges[i][0]]][arr[edges[i][1]]];
    }*/

    /* Unrolled version for K == 5 */
    result = adjacency_matrix[arr[0]][arr[1]] +
              adjacency_matrix[arr[0]][arr[2]] +
              adjacency_matrix[arr[0]][arr[3]] +
              adjacency_matrix[arr[0]][arr[4]] +
              adjacency_matrix[arr[1]][arr[2]] +
              adjacency_matrix[arr[1]][arr[3]] +
              adjacency_matrix[arr[1]][arr[4]] +
              adjacency_matrix[arr[2]][arr[3]] +
              adjacency_matrix[arr[2]][arr[4]] +
              adjacency_matrix[arr[3]][arr[4]];

    return result;
}

/*
 * Populates adjacency matrix based on contents of char array.
 */
void GetAdjacencyMatrixFromCharArray(char bit_arr[], int adj[N][N])
{
    int x = 0;

    for (int i = 0; i < N; i++) {
        for (int j = i + 1; j < N; j++) {
            adj[i][j] = bit_arr[x];
            adj[j][i] = bit_arr[x];
            x++;
        }
    }
}

/*
 * Grab an array of edges to check for cliques. These are represented 
 * as a tuple of indices into the array returned by getElement(m), e.g.:
 * 
 * {
 *  {0, 1},
 *  {0, 2},
 *  {1, 2}
 * }
 * 
 * For k = 3
 * 
 */
void GetEdgePossibilities(int arr[KC2][2])
{
    int num_edges = KC2;
    for (int i = 0; i < num_edges; i++) {
        GetElement(i, K, 2, arr[i]);
    }
}

/*
 *Populates arr with the mth lexicographic subset of size K from N vertices.
 */
void GetElement(int m, int arr[])
{
    int a = N;
    int b = K;
    int x = (Choose(N, K) - 1) - m; // x is the "dual" of m

    for (int i = 0; i < K; i++) {
        arr[i] = GetLargestV(a, b, x); //largest value v where v < a and vCb < x
        x = x - Choose(arr[i], b);
        a = arr[i];
        b--;
    }

    for (int i = 0; i < K; i++) {
        arr[i] = (N - 1) - arr[i];
    }
}

/*
 * Same as above, but allows specification of n and k.
 * Used by GetEdgePossibilities().
 */
void GetElement(int m, int n, int k, int arr[]) 
{
    int a = n;
    int b = k;
    int x = (Choose(n, k) - 1) - m; // x is the "dual" of m

    for (int i = 0; i < k; i++) {
        arr[i] = GetLargestV(a, b, x); //largest value v where v < a and vCb < x
        x = x - Choose(arr[i], b);
        a = arr[i];
        b--;
    }

    for (int i = 0; i < k; i++) {
        arr[i] = (n - 1) - arr[i];
    }
}

/*
 * Returns largest value v where v < a and vCb <= x
 */
int GetLargestV(int a, int b, int x)
{
    int v = a - 1;

    while (Choose(v, b) > x) {
        v--;
    }

    return v;
    
}

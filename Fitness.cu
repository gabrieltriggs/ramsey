/* Fitness function to evaluate the number of monochromatic 
 * cliques present in a given graph.
 *
 * Jon Johnson, Gabriel Triggs
 */
#include <iostream>
#include <stdlib.h>
#include <string>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include "Fitness.h"



//global variables
std::string bit_string = "011001110000011100100001110110000000010100110010010111110111010100111111110100011001000010011001111100101011111000001010011101101110100101011001100110001101101000000011000010110100011111001110100010101011001110010001110000111101000101010111100100000101110111101101011010000011000110001101001110110110111001011001101011110100011111011010001100010100010100101011110101100010001100101111011011000010110000101001001010101000101110110111110011101100001000100011111011101001010101111010101110011010001111000110111010011011001001011001100000000100110010111111100010010001011010110011010101001110010100111001001011100100100100100010110011110000100101010111110101101000001111001111101100011111001010101000010011001110100100011100101011000011100010101110011101111000101000110001010100100111100101111011010100001100010010101011000100010101110101010111101001110000110110101001000010011110111001100101111011100001010";
char *char_bits = new char[bit_string.size()];
int adjacency_matrix[N][N];

void print()
{
    std::cout << "test" << std::endl;
    std::cin >> "%d";
}

//int main(int argc, char *argv[])
//{
//    /* init char_bits */
//    for (int i = 0; i < bit_string.size(); i++) {
//        char_bits[i] = (bit_string[i] == '0') ? 0 : 1;
//    }
//
//    GetAdjacencyMatrixFromCharArray(char_bits, adjacency_matrix);
//
//    srand (time(NULL)); // init random seed
//    int upper_bound = choose_cache[N][K];
//    std::cout << upper_bound << std::endl;
//    int arr[5] = { 0, 0, 0, 0, 0 };
//
//
//
// //   // need to figure out time stuff
// //   long total_time = 0;
// //   long first_time = time(NULL); // get current time
// //   long time_limit = 5;
//  //int count = 0;
// //   do {
// //       int num_cliques = 0;
//
// //       /* evaluate every possible clique */
// //       for (int i = 0; i < upper_bound; i++) {
// //           GetElement(i, arr);
// //           
// //           int result = EvaluateEdges(arr);
// //           
// //           if (result == 0 || result == KC2) {
// //               num_cliques++;
// //               //print();
// //           }
// //       }
//
// //       long current_time = time(NULL);
// //       total_time = current_time - first_time;
// //       
// //       std::cout << "Cliques: " << num_cliques << std::endl;
//
// //       if (num_cliques < 10) {
// //           /* not going to happen */
// //           for (int j = 0; j < bit_string.size(); j++) {
// //               std::cout << char_bits[j];
// //           }
// //           std::cout << std::endl;
// //       }
//
// //       /* flip a random bit */
// //       int x = rand() % bit_string.size(); // random number over [0, char_bits.length)
// //       char_bits[x] = !char_bits[x];
// //       GetAdjacencyMatrixFromCharArray(char_bits, adjacency_matrix);
//  //  count++;
// //   } while (total_time < 5);
//  //std::cout << "Number of graphs processed: " << count << std::endl;
//  //std::cout << "Time per graph: " << total_time / (float)count << std::endl;
//  std::getchar();
//}

/*
 * Returns the number of edges in arr that are "red" (1).
 *
 * If this return KC2, it is a red clique.
 * If this returns 0, it is a blue clique.
 */
int EvaluateEdges(int arr[])
{
    return adjacency_matrix[arr[0]][arr[1]] +
           adjacency_matrix[arr[0]][arr[2]] +
           adjacency_matrix[arr[0]][arr[3]] +
           adjacency_matrix[arr[0]][arr[4]] +
           adjacency_matrix[arr[1]][arr[2]] +
           adjacency_matrix[arr[1]][arr[3]] +
           adjacency_matrix[arr[1]][arr[4]] +
           adjacency_matrix[arr[2]][arr[3]] +
           adjacency_matrix[arr[2]][arr[4]] +
           adjacency_matrix[arr[3]][arr[4]];
}

/*
 * Populates adjacency matrix based on contents of char array.
 */
void GetAdjacencyMatrixFromCharArray(char bit_arr[], int adj[N][N])
{
    int x = 0;

    for (int i = 0; i < N; i++) {
        for (int j = i + 1; j < N; j++) {
            adj[i][j] = bit_arr[x];
            adj[j][i] = bit_arr[x];
            x++;
        }
    }
}

/*
 *Populates arr with the mth lexicographic subset of size K from N vertices.
 */
void GetElement(int m, int arr[])
{
    int a = N;
    int b = K;
    int x = (choose_cache[N][K] - 1) - m; // x is the "dual" of m

    for (int i = 0; i < K; i++) {
        arr[i] = GetLargestV(a, b, x); //largest value v where v < a and vCb < x
        x = x - choose_cache[arr[i]][b];
        a = arr[i];
        b--;
    }

    for (int i = 0; i < K; i++) {
        arr[i] = (N - 1) - arr[i];
    }
}

/*
 * Returns largest value v where v < a and vCb <= x
 */
int GetLargestV(int a, int b, int x)
{
    int v = a - 1;

    while (choose_cache[v][b] > x) {
        v--;
    }

    return v;
    
}


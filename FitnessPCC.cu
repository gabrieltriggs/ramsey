#include "hip/hip_runtime.h"
/*
 * Fitness function to evaluate the number of monochromatic 
 * cliques present in a given graph.
 *
 * Jon Johnson, Gabriel Triggs
 */
#include <iostream>
#include <stdlib.h>
#include <string>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#define N 43
#define K 5
#define KC2 10

//fuction declarations
int Choose(int, int);
int EvaluateEdges(int[]);
void GetAdjacencyMatrixFromCharArray(char[], int[N][N]);
void GetEdgePossibilities(int[KC2][2]);
void GetElement(int, int[]);
void GetElement(int, int, int, int[]);
int GetLargestV(int, int, int);

//global variables
int edges[KC2][2];
std::string bit_string = "011001110000011100100001110110000000010100110010010111110111010100111111110100011001+000010011001111100101011111000001010011101101110100101011001100110001101101000000011000010110100011111001110100010101011001110010001110000111101000101010111100100000101110111101101011010000011000110001101001110110110111001011001101011110100011111011010001100010100010100101011110101100010001100101111011011000010110000101001001010101000101110110111110011101100001000100011111011101001010101111010101110011010001111000110111010011011001001011001100000000100110010111111100010010001011010110011010101001110010100111001001011100100100100100010110011110000100101010111110101101000001111001111101100011111001010101000010011001110100100011100101011000011100010101110011101111000101000110001010100100111100101111011010100001100010010101011000100010101110101010111101001110000110110101001000010011110111001100101111011100001010";
char *char_bits = new char[bit_string.size()];
int adjacency_matrix[N][N];
int choose_cache[][6] = {
        {0, 0, 0, 0, 0, 0},
        {0, 1, 0, 0, 0, 0},
        {0, 2, 1, 0, 0, 0},
        {0, 3, 3, 1, 0, 0},
        {0, 4, 6, 4, 1, 0},
        {0, 5, 10, 10, 5, 1},
        {0, 6, 15, 20, 15, 6},
        {0, 7, 21, 35, 35, 21},
        {0, 8, 28, 56, 70, 56},
        {0, 9, 36, 84, 126, 126},
        {0, 10, 45, 120, 210, 252},
        {0, 11, 55, 165, 330, 462},
        {0, 12, 66, 220, 495, 792},
        {0, 13, 78, 286, 715, 1287},
        {0, 14, 91, 364, 1001, 2002},
        {0, 15, 105, 455, 1365, 3003},
        {0, 16, 120, 560, 1820, 4368},
        {0, 17, 136, 680, 2380, 6188},
        {0, 18, 153, 816, 3060, 8568},
        {0, 19, 171, 969, 3876, 11628},
        {0, 20, 190, 1140, 4845, 15504},
        {0, 21, 210, 1330, 5985, 20349},
        {0, 22, 231, 1540, 7315, 26334},
        {0, 23, 253, 1771, 8855, 33649},
        {0, 24, 276, 2024, 10626, 42504},
        {0, 25, 300, 2300, 12650, 53130},
        {0, 26, 325, 2600, 14950, 65780},
        {0, 27, 351, 2925, 17550, 80730},
        {0, 28, 378, 3276, 20475, 98280},
        {0, 29, 406, 3654, 23751, 118755},
        {0, 30, 435, 4060, 27405, 142506},
        {0, 31, 465, 4495, 31465, 169911},
        {0, 32, 496, 4960, 35960, 201376},
        {0, 33, 528, 5456, 40920, 237336},
        {0, 34, 561, 5984, 46376, 278256},
        {0, 35, 595, 6545, 52360, 324632},
        {0, 36, 630, 7140, 58905, 376992},
        {0, 37, 666, 7770, 66045, 435897},
        {0, 38, 703, 8436, 73815, 501942},
        {0, 0, 741, 9139, 82251, 575757},
        {0, 0, 0, 9880, 91390, 658008},
        {0, 0, 0, 0, 101270, 749398},
        {0, 0, 0, 0, 0, 850668},
        {0, 0, 0, 0, 0, 962598}};

void print()
{
    std::cout << "test" << std::endl;
    std::cin >> "%d";
}

int main(int argc, char *argv[])
{
    /* init char_bits */
    for (int i = 0; i < bit_string.size(); i++) {
        char_bits[i] = (bit_string[i] == '0') ? 0 : 1;
    }

    GetEdgePossibilities(edges);
    GetAdjacencyMatrixFromCharArray(char_bits, adjacency_matrix);

    srand (time(NULL)); // init random seed
    int upper_bound = Choose(N, K);
    std::cout << upper_bound << std::endl;
    int arr[5] = { 0, 0, 0, 0, 0 };

    // need to figure out time stuff
    //long total_time = 0;
    //long first_time = time(NULL); // get current time
    //long time_limit = 5;
    //print();

    do {
        int num_cliques = 0;

        /* evaluate every possible clique */
        for (int i = 0; i < upper_bound; i++) {
            GetElement(i, arr);
            
            int result = EvaluateEdges(arr);
            
            if (result == 0 || result == KC2) {
                num_cliques++;
                //print();
            }
        }

        //long current_time = time(NULL);
        //total_time = current_time - first_time;
        
        std::cout << "Cliques: " << num_cliques << std::endl;

        if (num_cliques < 10) {
            /* not going to happen */
            for (int j = 0; j < bit_string.size(); j++) {
                std::cout << char_bits[j];
            }
            std::cout << std::endl;
        }

        /* flip a random bit */
        int x = rand() % bit_string.size(); // random number over [0, char_bits.length)
        char_bits[x] = !char_bits[x];
        GetAdjacencyMatrixFromCharArray(char_bits, adjacency_matrix);
    } while (1); //FIX THIS FIX IT FIX ITASDFGFJASDFIFAJSDIFJASWDIFJASDIFJASCIFAJSIFAJSDKIFAHJSDIFNHJASDIFNHASDIFNHASDIFHASDIHFASKIDNHFASDKINHF
}

/*
 * Returns nCk.
 */
int Choose(int n, int k)
{
    return choose_cache[n][k];
//  if (n < k) {
//      return 0;
//  } else if (n == k) {
//      return 1;
//  }
//
//  /* check to see whether it's already cached */
//  if (choose_cache[n-1][k-1] != 0) {
//      return choose_cache[n-1][k-1];
//  }
//
//  /* take advantage of the fact that nCk == nC(n-k) to do faster computation */
//  int diff;
//  int max;
//
//  if (k < (n - k)) {
//      diff = n - k;
//      max = k;
//  } else {
//      diff = k;
//      max = n - k;
//  }
//
//  int ans = diff + 1;
//
//  for (int i = 2; i <= max; i++) {
//      ans = (ans * (diff + i)) / i;
//  }
//
//  /* cache answer before returning */
//  choose_cache[n-1][k-1] = ans;
//
//  return ans;
}

/*
 * Returns the number of edges in arr that are "red" (1).
 *
 * If this return KC2, it is a red clique.
 * If this returns 0, it is a blue clique.
 */
int EvaluateEdges(int arr[])
{
    int result = 0;

    /*for (int i = 0; i < KC2; i++) {
        result += adjacency_matrix[arr[edges[i][0]]][arr[edges[i][1]]];
    }*/

    /* Unrolled version for K == 5 */
    result = adjacency_matrix[arr[0]][arr[1]] +
              adjacency_matrix[arr[0]][arr[2]] +
              adjacency_matrix[arr[0]][arr[3]] +
              adjacency_matrix[arr[0]][arr[4]] +
              adjacency_matrix[arr[1]][arr[2]] +
              adjacency_matrix[arr[1]][arr[3]] +
              adjacency_matrix[arr[1]][arr[4]] +
              adjacency_matrix[arr[2]][arr[3]] +
              adjacency_matrix[arr[2]][arr[4]] +
              adjacency_matrix[arr[3]][arr[4]];

    return result;
}

/*
 * Populates adjacency matrix based on contents of char array.
 */
void GetAdjacencyMatrixFromCharArray(char bit_arr[], int adj[N][N])
{
    int x = 0;

    for (int i = 0; i < N; i++) {
        for (int j = i + 1; j < N; j++) {
            adj[i][j] = bit_arr[x];
            adj[j][i] = bit_arr[x];
            x++;
        }
    }
}

/*
 * Grab an array of edges to check for cliques. These are represented 
 * as a tuple of indices into the array returned by getElement(m), e.g.:
 * 
 * {
 *  {0, 1},
 *  {0, 2},
 *  {1, 2}
 * }
 * 
 * For k = 3
 * 
 */
void GetEdgePossibilities(int arr[KC2][2])
{
    int num_edges = KC2;
    for (int i = 0; i < num_edges; i++) {
        GetElement(i, K, 2, arr[i]);
    }
}

/*
 *Populates arr with the mth lexicographic subset of size K from N vertices.
 */
void GetElement(int m, int arr[])
{
    int a = N;
    int b = K;
    int x = (choose_cache[N][K] - 1) - m; // x is the "dual" of m

    for (int i = 0; i < K; i++) {
        arr[i] = GetLargestV(a, b, x); //largest value v where v < a and vCb < x
        x = x - choose_cache[arr[i]][b];
        a = arr[i];
        b--;
    }

    for (int i = 0; i < K; i++) {
        arr[i] = (N - 1) - arr[i];
    }
}

/*
 * Same as above, but allows specification of n and k.
 * Used by GetEdgePossibilities().
 */
void GetElement(int m, int n, int k, int arr[]) 
{
    int a = n;
    int b = k;
    int x = (Choose(n, k) - 1) - m; // x is the "dual" of m

    for (int i = 0; i < k; i++) {
        arr[i] = GetLargestV(a, b, x); //largest value v where v < a and vCb < x
        x = x - Choose(arr[i], b);
        a = arr[i];
        b--;
    }

    for (int i = 0; i < k; i++) {
        arr[i] = (n - 1) - arr[i];
    }
}

/*
 * Returns largest value v where v < a and vCb <= x
 */
int GetLargestV(int a, int b, int x)
{
    int v = a - 1;

    while (choose_cache[v][b] > x) {
        v--;
    }

    return v;
    
}
